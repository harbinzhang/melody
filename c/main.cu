
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <ctime>
#include <chrono>
#include <hipfft/hipfft.h>

#define BUFFER_SIZE 4096

// Complex data type
typedef float2 Complex;

#define SIGNAL_SIZE 100

typedef struct  WAV_HEADER
{
    /* RIFF Chunk Descriptor */
    uint8_t         RIFF[4];        // RIFF Header Magic header
    uint32_t        ChunkSize;      // RIFF Chunk Size
    uint8_t         WAVE[4];        // WAVE Header
    /* "fmt" sub-chunk */
    uint8_t         fmt[4];         // FMT header
    uint32_t        Subchunk1Size;  // Size of the fmt chunk
    uint16_t        AudioFormat;    // Audio format 1=PCM,6=mulaw,7=alaw,     257=IBM Mu-Law, 258=IBM A-Law, 259=ADPCM
    uint16_t        NumOfChan;      // Number of channels 1=Mono 2=Sterio
    uint32_t        SamplesPerSec;  // Sampling Frequency in Hz
    uint32_t        bytesPerSec;    // bytes per second
    uint16_t        blockAlign;     // 2=16-bit mono, 4=16-bit stereo
    uint16_t        bitsPerSample;  // Number of bits per sample
    /* "data" sub-chunk */
    uint8_t         Subchunk2ID[4]; // "data"  string
    uint32_t        Subchunk2Size;  // Sampled data length
} wav_hdr;
int getFileSize(FILE* inFile);



int main(int argc, char ** argv) {
    wav_hdr wavHeader;
    int headerSize = sizeof(wav_hdr);

    const char* filePath;
    filePath = argv[1];

    FILE* wavFile = fopen(filePath, "r");
    if (wavFile == nullptr)
    {
        fprintf(stderr, "Unable to open wave file: %s\n", filePath);
        return 1;
    }

    
    //Read the header
    size_t bytesRead = fread(&wavHeader, 1, headerSize, wavFile);
    float data_array[wavHeader.Subchunk2Size];
    if (bytesRead > 0)
    {

        //Read the data
        // uint16_t bytesPerSample = wavHeader.bitsPerSample / 8;      //Number     of bytes per sample
        // uint64_t numSamples = wavHeader.ChunkSize / bytesPerSample; //How many samples are in the wav file?
        float* buffer = new float[BUFFER_SIZE];

        int i = 0;
        while ((bytesRead = fread(buffer, sizeof buffer[0], BUFFER_SIZE / (sizeof buffer[0]), wavFile)) > 0)
        {
            /** DO SOMETHING WITH THE WAVE DATA HERE **/
            memcpy(&data_array[BUFFER_SIZE*i], &buffer[0], bytesRead);
            i++;
        }
        delete [] buffer;
        buffer = nullptr;
        printf("%d\n", i);

    }
    fclose(wavFile);

    printf("[simpleCUFFT] is starting...\n");
    // Allocate host memory for the signal
    // Complex* h_signal = (Complex*)malloc(sizeof(Complex) * SIGNAL_SIZE);
    float* h_signal = (float*) malloc(sizeof(float) * SIGNAL_SIZE);
    memcpy(h_signal, &data_array[0], SIGNAL_SIZE);

    for(int i = 0; i < SIGNAL_SIZE; i++){
        printf("%f\n", h_signal[i]);
    }

    // Initalize the memory for the signal
    int mem_size = sizeof(float) * SIGNAL_SIZE;

    // Allocate device memory for signal
    float* g_signal;
    hipMalloc((void**)&g_signal, mem_size);
    // Copy host memory to device
    hipMemcpy(g_signal, h_signal, mem_size,
               hipMemcpyHostToDevice);

    Complex* g_out;
    hipMalloc((void**)&g_out, sizeof(Complex) * SIGNAL_SIZE);

    // CUFFT plan
    hipfftHandle plan;
    hipfftPlan1d(&plan, SIGNAL_SIZE, HIPFFT_R2C, 1);

    // Transform signal and kernel
    printf("Transforming signal cufftExecC2C\n");
    hipfftExecR2C(plan, (float *)g_signal, (Complex *)g_out);    

    
    // Transform signal back
    printf("Transforming signal back cufftExecC2C\n");
    hipfftExecC2R(plan, (Complex *)g_out, (float *)g_signal);


    float* h_out = h_signal;
    hipMemcpy(h_out, g_signal, mem_size, hipMemcpyDeviceToHost);


    for(int i = 0; i < SIGNAL_SIZE; i++){
        printf("%f\n", h_out[i]);
    }

    hipfftDestroy(plan);

    free(h_signal);

    hipFree(g_signal);
    hipFree(g_out);

    return 0;
}

// find the file size
int getFileSize(FILE* inFile)
{
    int fileSize = 0;
    fseek(inFile, 0, SEEK_END);

    fileSize = ftell(inFile);

    fseek(inFile, 0, SEEK_SET);
    return fileSize;
}
